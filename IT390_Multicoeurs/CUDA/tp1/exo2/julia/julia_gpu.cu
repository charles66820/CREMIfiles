#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "common/book.h"
#include "common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel( unsigned char *ptr ) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = y * DIM + x;
    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

    //Allouer l'espace de dev_bitmap de taill bitmap.image_size()
    hipMalloc((void **)&dev_bitmap, bitmap.image_size());

    data.dev_bitmap = dev_bitmap;

    //Créer une grill 2D dont la taille de chaque dimension est DN
    dim3 grid(DIM, DIM);

    //Appel au kernel avec le nombre de blocks et dev_bitmap en paramétre
    kernel<<<grid, 1>>>(dev_bitmap);

    //Copier le résultat de dev_bitmap dans bitmap.get_ptr()
    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

    // Libérer l'espace allouée à dev_bit
    hipFree(dev_bitmap);

    bitmap.display_and_exit();
}

