#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <random>

#define N (1024 * 1024)

// __global__ void kernel(void) {}

__global__ void add(int *a, int *b, int *c, int n) {
  int indice = threadIdx.x + blockIdx.x * blockDim.x;

  if (indice < n) c[indice] = a[indice] + b[indice];
}

void random_ints(int *list, int size) {
  for (size_t i = 0; i < size; i++) {
    list[i] = rand() % 100;
  }
}

int main(void) {
  int *a, *b, *c;
  int *gpu_a, *gpu_b, *gpu_c;
  int size = N * sizeof(int);

  hipMalloc((void **)&gpu_a, size);
  hipMalloc((void **)&gpu_b, size);
  hipMalloc((void **)&gpu_c, size);

  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  random_ints(a, N);
  random_ints(b, N);

  hipMemcpy(gpu_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, b, size, hipMemcpyHostToDevice);

  add<<<1024, 1024>>>(gpu_a, gpu_b, gpu_c, N);

  hipMemcpy(c, gpu_c, size, hipMemcpyDeviceToHost);

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_c);

  for (int i = 0; i < N; i++) {
    printf("%d %d %d\n", a[i], b[i], c[i]);
  }

  free(a);
  free(b);
  free(c);
  return 0;
}